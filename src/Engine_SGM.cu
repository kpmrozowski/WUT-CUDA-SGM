#include "Engine_SGM.hpp"
#include "stdio.h"
#include "Census_transform.hpp"
#include "Matching_cost.hpp"

namespace sgm {

template <typename T, size_t MAX_DISPARITY>
class Engine_SGM<T, MAX_DISPARITY>::Impl {

private:
	CensusTransform<T> m_census_left;
	CensusTransform<T> m_census_right;
	MatchingCost<MAX_DISPARITY> m_matching_cost;

public:
	Impl()
		: m_census_left()
		, m_census_right()
		, m_matching_cost()
	{ }

	void enqueue() {}
	void enqueue(
		output_type *dest_left,
		output_type *dest_right,
		const input_type *src_left,
		const input_type *src_right,
		int width,
		int height,
		const Parameters& param,
		hipStream_t stream)
	{
		printf("Stereo starts\n");
		m_census_left.enqueue(src_left, width, height, stream);
		m_census_right.enqueue(src_right, width, height, stream);
		m_matching_cost.enqueue(
			m_census_left.get_output(), 
			m_census_right.get_output(), 
			width, height);
		printf("Stereo ends\n");
	}

};


template <typename T, size_t MAX_DISPARITY>
Engine_SGM<T, MAX_DISPARITY>::Engine_SGM()
	: m_impl(new Impl())
{ }

template <typename T, size_t MAX_DISPARITY>
Engine_SGM<T, MAX_DISPARITY>::~Engine_SGM() = default;


template <typename T, size_t MAX_DISPARITY>
void Engine_SGM<T, MAX_DISPARITY>::execute()
{
	m_impl->enqueue();
	hipStreamSynchronize(0);
}

template <typename T, size_t MAX_DISPARITY>
void Engine_SGM<T, MAX_DISPARITY>::execute(
	output_type *dest_left,
	output_type *dest_right,
	const input_type *src_left,
	const input_type *src_right,
	int width,
	int height,
	const Parameters& param)
{
	m_impl->enqueue(
		dest_left, dest_right,
		src_left, src_right,
		width, height,
		param,
		0);
	hipStreamSynchronize(0);
}


template class Engine_SGM<uint8_t,   64>;
template class Engine_SGM<uint8_t,  128>;
template class Engine_SGM<uint8_t,  256>;
template class Engine_SGM<uint16_t,  64>;
template class Engine_SGM<uint16_t, 128>;
template class Engine_SGM<uint16_t, 256>;

}

#include "hip/hip_runtime.h"
#include "Energy.hpp"

namespace sgm {

template <int DIRECTION, unsigned int MAX_DISPARITY>
__global__ void aggregate_vertical_path_kernel(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp)
{
	;
}

template <unsigned int MAX_DISPARITY>
void compute_energy_up2down(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	const int gdim = 1;
	const int bdim = width;
	aggregate_vertical_path_kernel<1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, cost_in, width, height, p1, p2, min_disp);
}

template <unsigned int MAX_DISPARITY>
void compute_energy_down2up(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	const int gdim = 1;
	const int bdim = width;
	aggregate_vertical_path_kernel<-1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, cost_in, width, height, p1, p2, min_disp);
}

template void compute_energy_up2down<64u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_up2down<128u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_up2down<256u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_down2up<64u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_down2up<128u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_down2up<256u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

}



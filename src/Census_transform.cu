#include "hip/hip_runtime.h"
#include <cstdio>
#include <Parameters.hpp>
#include "Census_transform.hpp"

namespace sgm {

namespace {

/**
 * @see https://www.spiedigitallibrary.org/journals/optical-engineering/volume-55/issue-06/063107/Improved-census-transform-for-noise-robust-stereo-matching/10.1117/1.OE.55.6.063107.full?SSO=1
 */
template <typename T>
__global__ void census_transform_kernel(
	feature_type *dest,
	const T *src,
	int width,
	int height
#ifdef DEBUG
	, bool *flag
	, int* bit0
#endif
	)
{
	const int padX = CESUS_WINDOW_WIDTH / 2;
	const int padY = CESUS_WINDOW_HEIGHT / 2;
	int win_x_max = padX;
	const int x = threadIdx.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	feature_type f = 0;
	if ( (padX <= x && x < width - padX) && (padY <= y && y < height - padY) ) {
		const T center = src[x + y * width];
        int bitPos0 = 0;
		int bitPos1 = CESUS_WINDOW_WIDTH * CESUS_WINDOW_HEIGHT - 1;
		for (int win_y = 0; win_y <= padY; ++win_y) {
			if (padY == win_y) win_x_max = -1;
			for (int win_x = -padX; win_x <= win_x_max; ++win_x, ++bitPos0, --bitPos1) {
				const T srcVal0 = src[ x + win_x + (y + win_y) * width ];
				const T srcVal1 = src[ x - win_x + (y - win_y) * width ];
				f |= (center < srcVal0) << bitPos0;
				f |= (center < srcVal1) << bitPos1;
			}
		}
#ifdef DEBUG
		if (bitPos0 == 22) *flag = true;
		*bit0 = bitPos0;
#endif
		dest[x + y * width] = f;
	}
	// else if ( (0 <= x && x < padX || width - padX <= x && x < width) 
	// 	   || (0 <= y && y < padY || height -padY <= y && y < height) ) {
	// 		dest[x + y * width] = 0;
	// }
}

int nextPowerOf2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

template <typename T>
void compute_census_transform(
	feature_type *dest,
	const T *src,
	int width,
	int height)
{
	printf("My cesus transform\n");
    int num_threads = 1024;
	int block_x_dim = nextPowerOf2(width);
	int block_y_dim = num_threads / block_x_dim;
	int grid_dim = height / block_y_dim;
	if (grid_dim * block_y_dim < height) {
		grid_dim += 1;
	}
	const dim3 bdim(block_x_dim, block_y_dim);
	const dim3 gdim(1, grid_dim, 1);
#ifdef DEBUG
	auto flag = DeviceBuffer<bool>(1);
	auto bit0 = DeviceBuffer<int>(1);
	bool flag_cpu[1] = {false};
	int bit0_cpu[1] = {0};
	hipMemcpy(flag.mutable_data(), flag_cpu, sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(bit0.mutable_data(), bit0_cpu, sizeof(int), hipMemcpyHostToDevice);
	census_transform_kernel<<<gdim, bdim, 0>>>(dest, src, width, height, flag.mutable_data(), bit0.mutable_data());
	feature_type lookup[height * width];
	printf("dest.size()=%zd\n", sizeof(lookup)/sizeof(lookup[0]));
	hipMemcpy(lookup, dest, sizeof(feature_type) * height * width, hipMemcpyDeviceToHost);
	printf("dest[90][330]=%ld\n", lookup[90 + 330 * width]);
	hipMemcpy(flag_cpu, flag.data(), sizeof(bool), hipMemcpyDeviceToHost);
	if (*flag_cpu) printf("fatal error\n");
	hipMemcpy(bit0_cpu, bit0.data(), sizeof(int), hipMemcpyDeviceToHost);
	printf("bit0=%d\n", *bit0_cpu);
#else
	census_transform_kernel<<<gdim, bdim, 0>>>(dest, src, width, height);
#endif
}

}


template <typename T>
CensusTransform<T>::CensusTransform()
	: m_feature_buffer()
{ }

template <typename T>
void CensusTransform<T>::compute(
	const T *src,
	int width,
	int height)
{
	if(m_feature_buffer.size() != static_cast<size_t>(width * height)){
		m_feature_buffer = DeviceBuffer<feature_type>(width * height);
	}
	compute_census_transform(
		m_feature_buffer.mutable_data(), src, width, height);
}

template class CensusTransform<uint8_t>;
template class CensusTransform<uint16_t>;

}

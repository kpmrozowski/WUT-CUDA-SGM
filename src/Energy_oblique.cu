#include "hip/hip_runtime.h"
#include "Energy.hpp"

namespace sgm {

template <int X_DIRECTION, int Y_DIRECTION, unsigned int MAX_DISPARITY>
__global__ void aggregate_oblique_path_kernel(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp)
{
	;
}

template <unsigned int MAX_DISPARITY>
void compute_energy_upL2downR(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	const int gdim = 1;
	const int bdim = width;
	aggregate_oblique_path_kernel<1, 1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, cost_in, width, height, p1, p2, min_disp);
}

template <unsigned int MAX_DISPARITY>
void compute_energy_upR2downL(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	const int gdim = 1;
	const int bdim = width;
	aggregate_oblique_path_kernel<-1, 1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, cost_in, width, height, p1, p2, min_disp);
}

template <unsigned int MAX_DISPARITY>
void compute_energy_downR2upL(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	const int gdim = 1;
	const int bdim = width;
	aggregate_oblique_path_kernel<-1, -1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, cost_in, width, height, p1, p2, min_disp);
}

template <unsigned int MAX_DISPARITY>
void compute_energy_downL2upR(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream)
{
	const int gdim = 1;
	const int bdim = width;
	aggregate_oblique_path_kernel<1, -1, MAX_DISPARITY><<<gdim, bdim, 0, stream>>>(
		dest, cost_in, width, height, p1, p2, min_disp);
}


template void compute_energy_upL2downR<64u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_upL2downR<128u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_upL2downR<256u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_upR2downL<64u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_upR2downL<128u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_upR2downL<256u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_downR2upL<64u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_downR2upL<128u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_downR2upL<256u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_downL2upR<64u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_downL2upR<128u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

template void compute_energy_downL2upR<256u>(
	cost_type *dest,
	const cost_type *cost_in,
	int width,
	int height,
	unsigned int p1,
	unsigned int p2,
	int min_disp,
	hipStream_t stream);

}